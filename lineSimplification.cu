#include "hip/hip_runtime.h"
#include <iostream>
#include <sstream>
#include <fstream>
#include <vector>
#include <string>

/**
 * Input: trajectory of points, in order from first to last, list of points that cannot be simplified out
 * Output: list of simplified points
 */


__global__ void lsKernel(int* pointData, int arraySize) {
    
}

int myRandInt() {
    int num = rand() % 100 +1;
    if( rand( ) % 2 == 0 )
        num *= -1;
    return num;
}

int main(int argc, char * argv[]) {
    if (argc != 3) {
        cout << "Program requires 3 arguments: [exe] [trajectory] [point list]\n";
    }
    else {
        
    }
    

    // Create an array of numbers in traditional memory.
    // Fill the array with random values
    std::vector<int> data;
    srand(0);
    for( int i = 0; i < total; i++ )
    {
        int tmp = myRandInt( );
        data.push_back( tmp );
    }

    // Step 1:  Create an array on the GPU to hold the numbers that we
    // will do the sum3 computation on
    int *device_nums;
    hipMalloc( &device_nums, data.size() * sizeof( int) );

    // Step 2: Copy the data to the device array
    hipMemcpy(device_nums, &(data[0]), data.size() * sizeof( int), hipMemcpyHostToDevice);

    //Step 3:  We must keep track of the number of triples that sum
    // to 0.  We will create a single memory location (variable) on
    // the GPU that is SHARED among ALL threads.  Whenever a thread
    // finds a triple that sums to 0, this variable will be incremented
    int* device_count;
    hipMalloc( &device_count, sizeof( int) );
    {
        // initialize the count to 0
        int startCount = 0;
        hipMemcpy(device_count, &(startCount), sizeof( int), hipMemcpyHostToDevice);
    }


    // Just some code to time the kernel
    hipEvent_t startTotal, stopTotal;
    float timeTotal;
    hipEventCreate(&startTotal);
    hipEventCreate(&stopTotal);
    hipEventRecord( startTotal, 0 );



    // Step 4:  Decide how many threads we will organize into a block.  The
    // number of threads required will depend on the length of the array
    // containing random numbers.  Here, we are simply figuring out
    // how many threads we need based on the size of that array
    // (we allocated the array as an STL vector)
    //
    // Since EACH thread gets 2 fixed values, we are going to give threads
    // ID numbers that will indicate the array indexes of the 3 values
    // that will be fixed in that thread.  So, we create a 2 dimensional
    // thread block.  It simply labels each thread with 2 numbers that form
    // its identifier.
    dim3 threadsPerBlock(16,32);
    dim3 numBlocks((data.size() +threadsPerBlock.x-1)/ threadsPerBlock.x,
            (data.size() +threadsPerBlock.y-1)/ threadsPerBlock.y);

    std::cerr <<"data size: " <<(data.size()) << std::endl;
    std::cerr <<"block sizes: " <<(data.size() +threadsPerBlock.x-1)/ threadsPerBlock.x
        <<", " <<(data.size() +threadsPerBlock.y-1)/ threadsPerBlock.y <<  std::endl;


    // Step 5.  Now we have computed how many threads to launch.  We have
    // given each thread and identifier consisting of a pair (x,y).
    // Finally, launch the threads.
    sum3Kernel<<< numBlocks, threadsPerBlock>>> ( device_nums, data.size(), device_count );


    // Step 6:  After the threads have all finished, the count of triples that
    // sum to 0 is still stored on the GPU.  We just need to transfer it
    // back to the CPU so we can print it out.
    int totalFound;
    hipMemcpy(&totalFound, device_count, sizeof( int), hipMemcpyDeviceToHost);

    // stop the timer
    hipEventRecord( stopTotal, 0 );
    hipEventSynchronize( stopTotal );
    hipEventElapsedTime( &timeTotal, startTotal, stopTotal );
    hipEventDestroy( startTotal );
    hipEventDestroy( stopTotal );

    // print it out!
    std::cerr << "total time in seconds: " << timeTotal / 1000.0 << std::endl;
    std::cerr << "Total triples found: " << totalFound  <<std::endl;
}